#include "hip/hip_runtime.h"
/**
 * Copyright 1993-2012 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 */
#include <stdio.h>
#include <stdlib.h>
#include <iostream>

#include <hip/hip_runtime.h>


// Thread block size
#define BLOCK_SIZE 16

// Matrix multiplication kernel called by MatMul()
__global__ void mergesort_kernel(int n, int* v) {
	int p, r;
	int b = 1;
	while (b < n) {
		p = 0;
		while (p + b < n) {
			r = p + 2 * b;
			if (r > n)
				r = n;
			intercala(p, p + b, r, v);
			p = p + 2 * b;
		}
		b = 2 * b;
	}
}

void print(Matrix X) {
	for (int i = 0; i < X.height; i++) {
		for (int j = 0; j < X.width; j++) {
			std::cout << X.elements[i * X.width + j] << " ";
		}
		std::cout << "\n";
	}
}

// Matrix multiplication - Host code
// Matrix dimensions are assumed to be multiples of BLOCK_SIZE
void mergesort(int n, int* v) {
// Load A and B to device memory
	Matrix d_A;
	d_A.width = d_A.stride = A.width;
	d_A.height = A.height;
	size_t size = A.width * A.height * sizeof(float);
	hipMalloc(&d_A.elements, size);
	hipMemcpy(d_A.elements, A.elements, size, hipMemcpyHostToDevice);
	Matrix d_B;
	d_B.width = d_B.stride = B.width;
	d_B.height = B.height;
	size = B.width * B.height * sizeof(float);

	hipMalloc(&d_B.elements, size);
	hipMemcpy(d_B.elements, B.elements, size, hipMemcpyHostToDevice);
// Allocate C in device memory
	Matrix d_C;
	d_C.width = d_C.stride = C.width;
	d_C.height = C.height;
	size = C.width * C.height * sizeof(float);
	hipMalloc(&d_C.elements, size);
// Invoke kernel
	dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
	dim3 dimGrid(B.width / dimBlock.x, A.height / dimBlock.y);

	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	hipEventRecord(start);
	MatMulKernel<<<dimGrid, dimBlock>>>(d_A, d_B, d_C);
	hipEventRecord(stop);

	if (ELAPSED_TIME == 1) {
		hipEventSynchronize(stop);
		float milliseconds = 0;
		hipEventElapsedTime(&milliseconds, start, stop);
		std::cout << milliseconds << "\n";
	} else {
		print(C);
	}

// Read C from device memory
	hipMemcpy(C.elements, d_C.elements, size, hipMemcpyDeviceToHost);

// Free device memory
	hipFree(d_A.elements);
	hipFree(d_B.elements);
	hipFree(d_C.elements);
}

int main() {

	int n, m, q;
	n = m = q = 32;

	Matrix A;
	Matrix B;
	Matrix C;

	int sizeA = n * m * sizeof(float);
	A.height = n;
	A.width = m;
	A.elements = new float[sizeA];

	int sizeB = m * q * sizeof(float);
	B.height = m;
	B.width = q;
	B.elements = new float[sizeB];

	int sizeC = n * q * sizeof(float);
	C.height = n;
	C.width = q;
	C.elements = new float[sizeC];

	srand(time(NULL));
	for (int i = 0; i < A.height * A.width; i++) {
		A.elements[i] = rand() % 10;
	}

	for (int i = 0; i < B.height * B.width; i++) {
		B.elements[i] = rand() % 10;
	}

	//print(A);
	//printf("\n");
	//print(B);
	//printf("\n");

	mergesort(A, B, C);

	return 0;
}

