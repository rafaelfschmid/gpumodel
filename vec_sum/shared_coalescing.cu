#include "hip/hip_runtime.h"
/**
 * Copyright 1993-2012 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 */
#include <stdio.h>
#include <stdlib.h>
#include <iostream>

#include <hip/hip_runtime.h>

// Thread block size
//#ifndef BLOCK_SIZE
//#define BLOCK_SIZE 32
//#endif

__global__ void vec_sum(float *g_odata, float *g_idata, int n) {

	extern __shared__ float temp[]; // allocated on invocation

	int globalIndex = blockIdx.x * blockDim.x + threadIdx.x;
	int localIndex = threadIdx.x;

	temp[localIndex] = g_idata[globalIndex];
	__syncthreads();
	printf("temp[%d]=%f | global[%d]=%f\n", localIndex, temp[localIndex], globalIndex, temp[globalIndex]);

	for (int offset = BLOCK_SIZE/2; offset >= 1; offset /= 2) {
		if ( (localIndex < offset) && (globalIndex + offset < n) ) {
			//printf("block=%d | offset=%d | temp[%d]=%f | temp[%d]=%f\n", blockIdx.x, offset, localIndex, temp[localIndex], localIndex+offset, temp[localIndex+offset]);
			temp[localIndex] += temp[localIndex + offset];
		}
		__syncthreads();
	}

	if (localIndex == 0) {
		g_odata[blockIdx.x] = temp[localIndex]; // write output
		//printf("block=%d | temp[%d]=%f\n", blockIdx.x, localIndex, temp[localIndex]);
	}
}

void print(float* x, const int n) {
	for (int i = 0; i < n; i++) {
		std::cout << x[i]; // << " ";
	}
	std::cout << "\n";
}

// Matrix multiplication - Host code
// Matrix dimensions are assumed to be multiples of BLOCK_SIZE
void VectorSum(float* odata, float* idata, const int n) {
	// Load A and B to device memory
	float* g_idata;
	float* g_odata;

	// Invoke kernel
	//dim3 dimBlock(BLOCK_SIZE);
	//dim3 dimGrid(n / BLOCK_SIZE, 1);
	int block = BLOCK_SIZE;
	int grid = (n-1) / block +1;
	//printf("grid=%d\n", grid);

	size_t size = n * sizeof(float);
	size_t block_size = grid * sizeof(float);
	hipMalloc(&g_idata, size);
	hipMalloc(&g_odata, block_size);

	hipMemcpy(g_idata, idata, size, hipMemcpyHostToDevice);

	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	hipEventRecord(start);
	vec_sum<<<grid, block, block>>>(g_odata, g_idata, n);
	for(int i = grid; i > 1; i/= block) {
		grid = (i-1)/block+1;
		hipDeviceSynchronize();
		//printf("i=%d\n", i);
		vec_sum<<<grid, block, block>>>(g_odata, g_odata, i);
	}
	hipEventRecord(stop);

	hipError_t errSync = hipGetLastError();
	hipError_t errAsync = hipDeviceSynchronize();
	if (errSync != hipSuccess)
		printf("4: Sync kernel error: %s\n", hipGetErrorString(errSync));
	if (errAsync != hipSuccess)
		printf("4: Async kernel error: %s\n", hipGetErrorString(errAsync));

	// Read C from device memory
	hipMemcpy(odata, g_odata, sizeof(float), hipMemcpyDeviceToHost);

	if (ELAPSED_TIME == 1) {
		hipEventSynchronize(stop);
		float milliseconds = 0;
		hipEventElapsedTime(&milliseconds, start, stop);
		std::cout << milliseconds << "\n";
	} else {
		//print(idata, n);
		print(odata, 1);
	}

// Free device memory
	hipFree(g_odata);
	hipFree(g_idata);
}

int main() {

	int n;
	scanf("%d", &n);

	float* idata, *odata;
	int size = n * sizeof(float);
	idata = new float[size];
	odata = new float[size];

	for (int i = 0; i < n; i++)
		scanf("%f", &idata[i]);

	VectorSum(odata, idata, n);

	//printf("result=%f\n", odata[n - 1]);

	free(odata);
	free(idata);

	return 0;
}
