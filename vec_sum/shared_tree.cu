#include "hip/hip_runtime.h"
/**
 * Copyright 1993-2012 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 */
#include <stdio.h>
#include <stdlib.h>
#include <iostream>

#include <hip/hip_runtime.h>

// Thread block size
#ifndef BLOCK_SIZE
#define BLOCK_SIZE 32
#endif

__global__ void vec_sum(float *g_odata, float *g_idata, int n) {
	extern __shared__ float temp[]; // allocated on invocation

	int globalIndex = blockIdx.x * blockDim.x + threadIdx.x;
	int localIndex = threadIdx.x;
	int offset = 1;

	if(2 * globalIndex + 1 < n) {
		temp[2 * localIndex] = g_idata[2 * globalIndex]; // load input into shared memory
		temp[2 * localIndex + 1] = g_idata[2 * globalIndex + 1];

		//for (int d = n >> 1; d > 0; d >>= 1) { // build sum in place up the tree
		for (int d = (2*BLOCK_SIZE) >> 1; d > 0; d >>= 1) { // build sum in place up the tree
			__syncthreads();
			if (localIndex < d) {
				int ai = offset * (2 * localIndex + 1) - 1;
				int bi = offset * (2 * localIndex + 2) - 1;
				temp[bi] += temp[ai];
			}
			offset *= 2;
		}

		if ((2*localIndex+1 == 2*BLOCK_SIZE - 1) || (2 * globalIndex + 1 == n-1)) {
			g_odata[blockIdx.x] = temp[2*localIndex+1]; // write output
			//printf("block=%d | temp[%d]=%f\n", blockIdx.x, 2*localIndex+1, temp[2*localIndex+1]);
		}
	}

}

void print(float* x, const int n) {
	for (int i = 0; i < n; i++) {
		std::cout << x[i] << " ";
	}
	std::cout << "\n\n";
}

// Matrix multiplication - Host code
// Matrix dimensions are assumed to be multiples of BLOCK_SIZE
void VectorSum(float* odata, float* idata, const int n) {
// Load A and B to device memory
	float* g_idata;
	float* g_odata;

	int block = BLOCK_SIZE;
	int grid = (n-1) / (2*block) +1;

	size_t size = n * sizeof(float);
	size_t block_size = size;//grid * sizeof(float);
	hipMalloc(&g_idata, size);
	hipMalloc(&g_odata, block_size);

	hipMemcpy(g_idata, idata, size, hipMemcpyHostToDevice);

	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	hipEventRecord(start);
	vec_sum<<<grid, block, 2*block>>>(g_odata, g_idata, n);
	for (int i = grid; i > 0; i /= (2*block)){
		grid = (i-1)/(2*block)+1;
		hipDeviceSynchronize();
		vec_sum<<<grid, block, 2*block>>>(g_odata, g_odata, i);
	}
	hipEventRecord(stop);

	hipError_t errSync = hipGetLastError();
	hipError_t errAsync = hipDeviceSynchronize();
	if (errSync != hipSuccess)
		printf("4: Sync kernel error: %s\n", hipGetErrorString(errSync));
	if (errAsync != hipSuccess)
		printf("4: Async kernel error: %s\n", hipGetErrorString(errAsync));

// Read C from device memory
	hipMemcpy(odata, g_odata, sizeof(float), hipMemcpyDeviceToHost);
	//hipMemcpy(odata, g_odata, size, hipMemcpyDeviceToHost);

	if (ELAPSED_TIME == 1) {
		hipEventSynchronize(stop);
		float milliseconds = 0;
		hipEventElapsedTime(&milliseconds, start, stop);
		std::cout << milliseconds << "\n";
	} else {
		//print(idata, n);
		print(odata, 1);
	}

// Free device memory
	hipFree(g_odata);
	hipFree(g_idata);
}

int main() {

	int n;
	scanf("%d", &n);

	float* idata, *odata;
	int size = n * sizeof(float);
	idata = new float[size];
	odata = new float[size];

	for (int i = 0; i < n; i++)
		scanf("%f", &idata[i]);

	VectorSum(odata, idata, n);

//printf("result=%f\n", odata[n - 1]);

	free(odata);
	free(idata);

	return 0;
}
