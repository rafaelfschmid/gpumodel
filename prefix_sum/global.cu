#include "hip/hip_runtime.h"
/**
 * Copyright 1993-2012 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 */
#include <stdio.h>
#include <stdlib.h>
#include <iostream>

#include <hip/hip_runtime.h>

// Thread block size
#ifndef BLOCK_SIZE
#define BLOCK_SIZE 512
#endif

__global__ void scan(float *g_odata, float *g_idata, int n) {

	extern __shared__ float temp[]; // allocated on invocation
	int thid = threadIdx.x;
	int pout = 0, pin = 1;
	// load input into shared memory.
	// This is exclusive scan, so shift right by one and set first elt to 0
	temp[pout * n + thid] = (thid > 0) ? g_idata[thid - 1] : 0;
	__syncthreads();
	for (int offset = 1; offset < n; offset *= 2) {
		pout = 1 - pout; // swap double buffer indices
		pin = 1 - pout;
		if (thid >= offset)
			temp[pout * n + thid] += temp[pin * n + thid - offset];
		else
			temp[pout * n + thid] = temp[pin * n + thid];
		__syncthreads();
	}
	g_odata[thid] = temp[pout * n + thid]; // write output
}

void print(float* x, const int n) {
	for (int i = 0; i < n; i++) {
		std::cout << x[i] << " ";
	}
	std::cout << "\n";
}

// Matrix multiplication - Host code
// Matrix dimensions are assumed to be multiples of BLOCK_SIZE
void PrefixSum(float* odata, float* idata, const int n) {
	// Load A and B to device memory
	float* g_idata;
	float* g_odata;

	size_t size = n * sizeof(float);
	hipMalloc(&g_idata, size);
	hipMalloc(&g_odata, size);

	hipMemcpy(g_idata, idata, size, hipMemcpyHostToDevice);

// Invoke kernel
	dim3 dimBlock(BLOCK_SIZE);
	//dim3 dimGrid(B.width / dimBlock.x, A.height / dimBlock.y);
	dim3 dimGrid(n/BLOCK_SIZE, 1);

	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	hipEventRecord(start);
	scan<<<dimGrid, dimBlock>>>(g_odata, g_idata, n);
	hipEventRecord(stop);

	hipError_t errSync = hipGetLastError();
	hipError_t errAsync = hipDeviceSynchronize();
	if (errSync != hipSuccess)
		printf("4: Sync kernel error: %s\n", hipGetErrorString(errSync));
	if (errAsync != hipSuccess)
		printf("4: Async kernel error: %s\n", hipGetErrorString(errAsync));

	// Read C from device memory
	hipMemcpy(odata, g_odata, size, hipMemcpyDeviceToHost);

	if (ELAPSED_TIME == 1) {
		hipEventSynchronize(stop);
		float milliseconds = 0;
		hipEventElapsedTime(&milliseconds, start, stop);
		std::cout << milliseconds << "\n";
	} else {
		print(odata, n);
	}

// Free device memory
	hipFree(g_odata);
	hipFree(g_idata);
}

int main() {

	int n;
	scanf("%d", &n);

	float* idata, *odata;
	int size = n * sizeof(float);
	idata = new float[size];
	odata = new float[size];

	for(int i = 0; i < n; i++)
		scanf("%f", &idata[i]);

	PrefixSum(odata, idata, n);

	free(odata);
	free(idata);

	return 0;
}
