#include "hip/hip_runtime.h"
/**
 * Copyright 1993-2012 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 */
#include <stdio.h>
#include <stdlib.h>
#include <iostream>

#include <hip/hip_runtime.h>

// Thread block size
#ifndef BLOCK_SIZE
#define BLOCK_SIZE 16
#endif

/*__global__ void scan(float *g_odata, float *g_idata, int n) {

	extern __shared__ float temp[]; // allocated on invocation

	int globalIndex = blockIdx.x * blockDim.x + threadIdx.x;
	int localIndex = threadIdx.x;

	temp[localIndex] = g_idata[globalIndex];
	__syncthreads();

	for (int offset = 1; offset < BLOCK_SIZE; offset *= 2) {
		if (localIndex >= offset)
			temp[localIndex] += temp[localIndex - offset];
		__syncthreads();
	}
	g_odata[globalIndex] = temp[localIndex]; // write output
}*/

__global__ void scan(float *g_odata, float *g_idata, int n) {

	extern __shared__ float temp[]; // allocated on invocation

	int globalIndex = blockIdx.x * blockDim.x + threadIdx.x;
	int localIndex = threadIdx.x;

	temp[localIndex] = g_idata[globalIndex];
	__syncthreads();

	for (int offset = 1; offset < BLOCK_SIZE; offset *= 2) {
		if (localIndex >= offset)
			temp[localIndex] += temp[localIndex - offset];
		__syncthreads();
	}

	g_idata[globalIndex] = temp[localIndex];

	if(localIndex == BLOCK_SIZE-1)
	{
		g_odata[blockIdx.x] = temp[localIndex]; // write output
		//printf("block=%d | temp[%d]=%f\n", blockIdx.x, localIndex, temp[localIndex]);
	}
}

__global__ void scan_block(float *g_odata, float *g_idata, int step, int n) {

	extern __shared__ float temp[]; // allocated on invocation

	//int globalIndex = blockIdx.x * blockDim.x + blockDim.x * (threadIdx.x + step) + blockDim.x - 1;
	int globalIndex = blockIdx.x * blockDim.x + blockDim.x * step + blockDim.x - 1;
	int localIndex = threadIdx.x;

	if(globalIndex < n-1) {
		//printf("global value=%d, global index=%d\n", g_odata[globalIndex], globalIndex);
		temp[localIndex] = g_odata[globalIndex];
		__syncthreads();

		for (int offset = 1; offset < BLOCK_SIZE; offset *= 2) {
			if (localIndex >= offset)
				temp[localIndex] += temp[localIndex - offset];
			__syncthreads();
		}

		g_idata[globalIndex] = temp[localIndex]; // write output
	}
}

__global__ void broadcast_sum(float *g_odata, float *g_idata, int step, int n) {

	extern __shared__ float temp[]; // allocated on invocation

	//int globalIndex = blockIdx.x * blockDim.x + blockDim.x * (threadIdx.x + step) + blockDim.x - 1;
	int globalIndex = blockIdx.x * blockDim.x + blockDim.x * step + blockDim.x - 1;
	int localIndex = threadIdx.x;

	if(globalIndex < n-1) {
		//printf("global value=%d, global index=%d\n", g_odata[globalIndex], globalIndex);
		temp[localIndex] = g_odata[globalIndex];
		__syncthreads();

		for (int offset = 1; offset < BLOCK_SIZE; offset *= 2) {
			if (localIndex >= offset)
				temp[localIndex] += temp[localIndex - offset];
			__syncthreads();
		}

		g_idata[globalIndex] = temp[localIndex]; // write output
	}
}

void print(float* x, const int n) {
	for (int i = 0; i < n; i++) {
		std::cout << x[i] << " ";
	}
	std::cout << "\n";
}

// Matrix multiplication - Host code
// Matrix dimensions are assumed to be multiples of BLOCK_SIZE
void PrefixSum(float* odata, float* idata, const int n) {
	// Load A and B to device memory
	float* g_idata;
	float* g_odata;

	// Invoke kernel
	//dim3 dimBlock(BLOCK_SIZE);
	//dim3 dimGrid(n / BLOCK_SIZE, 1);
	int block = BLOCK_SIZE;
	int grid = n / BLOCK_SIZE;

	size_t size = n * sizeof(float);
	size_t block_size = grid * sizeof(float);
	hipMalloc(&g_idata, size);
	hipMalloc(&g_odata, block_size);

	hipMemcpy(g_idata, idata, size, hipMemcpyHostToDevice);



	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	hipEventRecord(start);
	scan<<<grid, block, block>>>(g_odata, g_idata, n);

	hipError_t errSync = hipGetLastError();
	hipError_t errAsync = hipDeviceSynchronize();
	if (errSync != hipSuccess)
		printf("4: Sync kernel error: %s\n", hipGetErrorString(errSync));
	if (errAsync != hipSuccess)
		printf("4: Async kernel error: %s\n", hipGetErrorString(errAsync));

	//for (int step = 0; step < grid; step++) {
	/*scan_block<<<grid, block, block>>>(g_odata, g_idata, step, n);

	errSync = hipGetLastError();
	errAsync = hipDeviceSynchronize();
	if (errSync != hipSuccess)
		printf("4: Sync kernel error: %s\n", hipGetErrorString(errSync));
	if (errAsync != hipSuccess)
		printf("4: Async kernel error: %s\n", hipGetErrorString(errAsync));

	broadcast_sum<<<grid, block, block>>>(g_odata, g_idata, step, n);

	errSync = hipGetLastError();
	errAsync = hipDeviceSynchronize();
	if (errSync != hipSuccess)
		printf("4: Sync kernel error: %s\n", hipGetErrorString(errSync));
	if (errAsync != hipSuccess)
		printf("4: Async kernel error: %s\n", hipGetErrorString(errAsync));

	}*/
	hipEventRecord(stop);

	// Read C from device memory
	hipMemcpy(odata, g_odata, block_size, hipMemcpyDeviceToHost);

	if (ELAPSED_TIME == 1) {
		hipEventSynchronize(stop);
		float milliseconds = 0;
		hipEventElapsedTime(&milliseconds, start, stop);
		std::cout << milliseconds << "\n";
	} else {
		print(idata, n);
		print(odata, grid);
	}

// Free device memory
	hipFree(g_odata);
	hipFree(g_idata);
}

int main() {

	int n;
	scanf("%d", &n);

	float* idata, *odata;
	int size = n * sizeof(float);
	idata = new float[size];
	odata = new float[size];

	for (int i = 0; i < n; i++)
		scanf("%f", &idata[i]);

	print(idata, n);
	print(odata, n);

	PrefixSum(odata, idata, n);

	printf("result=%f\n", odata[n - 1]);

	free(odata);
	free(idata);

	return 0;
}
