#include "hip/hip_runtime.h"
/**
 * Copyright 1993-2012 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 */
#include <stdio.h>
#include <stdlib.h>
#include <iostream>

#include <hip/hip_runtime.h>

// Thread block size
#ifndef BLOCK_SIZE
#define BLOCK_SIZE 16
#endif

__global__ void scan(float *g_odata, float *g_idata, int n) {

	extern __shared__ float temp[]; // allocated on invocation

	int globalIndex = blockIdx.x*blockDim.x + threadIdx.x;
	int localIndex = threadIdx.x;

	temp[localIndex] = g_idata[globalIndex];
	__syncthreads();

	for (int offset = 1; offset < BLOCK_SIZE; offset *= 2) {
		if (localIndex >= offset)
			temp[localIndex] += temp[localIndex - offset];
		__syncthreads();
	}

	g_odata[globalIndex] = temp[localIndex]; // write output
}

__global__ void scan_block(float *g_odata, float *g_idata, int n) {

	extern __shared__ float temp[]; // allocated on invocation

	int globalIndex = blockIdx.x*blockDim.x + blockDim.x*threadIdx.x + blockDim.x;
	int localIndex = threadIdx.x;

	temp[localIndex] = g_idata[globalIndex];
	__syncthreads();

	for (int offset = 1; offset < BLOCK_SIZE; offset *= 2) {
		if (localIndex >= offset)
			temp[localIndex] += temp[localIndex - offset];
		__syncthreads();
	}

	g_odata[globalIndex] = temp[localIndex]; // write output
}


void print(float* x, const int n) {
	for (int i = 0; i < n; i++) {
		std::cout << x[i] << " ";
	}
	std::cout << "\n";
}

// Matrix multiplication - Host code
// Matrix dimensions are assumed to be multiples of BLOCK_SIZE
void PrefixSum(float* odata, float* idata, const int n) {
	// Load A and B to device memory
	float* g_idata;
	float* g_odata;

	size_t size = n * sizeof(float);
	hipMalloc(&g_idata, size);
	hipMalloc(&g_odata, size);

	hipMemcpy(g_idata, idata, size, hipMemcpyHostToDevice);

// Invoke kernel
	dim3 dimBlock(BLOCK_SIZE);
	dim3 dimGrid(n / BLOCK_SIZE, 1);

	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	hipEventRecord(start);
	scan<<<dimGrid, dimBlock, BLOCK_SIZE>>>(g_odata, g_idata, n);
	hipEventRecord(stop);

	hipError_t errSync = hipGetLastError();
	hipError_t errAsync = hipDeviceSynchronize();
	if (errSync != hipSuccess)
		printf("4: Sync kernel error: %s\n", hipGetErrorString(errSync));
	if (errAsync != hipSuccess)
		printf("4: Async kernel error: %s\n", hipGetErrorString(errAsync));

	// Read C from device memory
	hipMemcpy(odata, g_odata, size, hipMemcpyDeviceToHost);

	if (ELAPSED_TIME == 1) {
		hipEventSynchronize(stop);
		float milliseconds = 0;
		hipEventElapsedTime(&milliseconds, start, stop);
		std::cout << milliseconds << "\n";
	} else {
		print(idata, n);
		print(odata, n);
	}

// Free device memory
	hipFree(g_odata);
	hipFree(g_idata);
}

int main() {

	int n;
	scanf("%d", &n);

	float* idata, *odata;
	int size = n * sizeof(float);
	idata = new float[size];
	odata = new float[size];

	for (int i = 0; i < n; i++)
		scanf("%f", &idata[i]);

	print(idata, n);
	print(odata, n);

	PrefixSum(odata, idata, n);

	printf("result=%f\n", odata[n - 1]);

	free(odata);
	free(idata);

	return 0;
}
