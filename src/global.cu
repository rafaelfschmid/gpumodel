#include "hip/hip_runtime.h"
/**
 * Copyright 1993-2012 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 */
#include <stdio.h>
#include <stdlib.h>
#include <iostream>

#include <hip/hip_runtime.h>

// Matrices are stored in row-major order:
// M(row, col) = *(M.elements + row * M.width + col)
typedef struct {
	int width;
	int height;
	float* elements;
} Matrix;

// Thread block size
#define BLOCK_SIZE 16

// Forward declaration of the matrix multiplication kernel
__global__ void MatMulKernel(const Matrix, const Matrix, Matrix);

void print(Matrix X) {
	for (int i = 0; i < X.height; i++) {
		for (int j = 0; j < X.width; j++) {
			std::cout << X.elements[i * X.width + j] << " ";
		}
		std::cout << "\n";
	}
}

// Matrix multiplication - Host code
// Matrix dimensions are assumed to be multiples of BLOCK_SIZE
void MatMul(const Matrix A, const Matrix B, Matrix C) {
// Load A and B to device memory
	Matrix d_A;
	d_A.width = A.width;
	d_A.height = A.height;
	size_t size = A.width * A.height * sizeof(float);
	hipMalloc(&d_A.elements, size);
	hipMemcpy(d_A.elements, A.elements, size, hipMemcpyHostToDevice);
	Matrix d_B;
	d_B.width = B.width;
	d_B.height = B.height;
	size = B.width * B.height * sizeof(float);
	hipMalloc(&d_B.elements, size);
	hipMemcpy(d_B.elements, B.elements, size, hipMemcpyHostToDevice);
// Allocate C in device memory
	Matrix d_C;
	d_C.width = C.width;
	d_C.height = C.height;
	size = C.width * C.height * sizeof(float);
	hipMalloc(&d_C.elements, size);
// Invoke kernel
	dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
	dim3 dimGrid(B.width / dimBlock.x, A.height / dimBlock.y);

	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	hipEventRecord(start);
	MatMulKernel<<<dimGrid, dimBlock>>>(d_A, d_B, d_C);
	hipEventRecord(stop);

	if (ELAPSED_TIME == 1) {
		hipEventSynchronize (stop);
		float milliseconds = 0;
		hipEventElapsedTime(&milliseconds, start, stop);
		std::cout << milliseconds << "\n";
	}
	else {
		print(C);
	}

// Read C from device memory
	hipMemcpy(C.elements, d_C.elements, size, hipMemcpyDeviceToHost);

// Free device memory
	hipFree(d_A.elements);
	hipFree(d_B.elements);
	hipFree(d_C.elements);
}

int main() {

	int n, m, q;
	scanf("%d", &n);
	m = q = n;

	Matrix A;
	Matrix B;
	Matrix C;

	int sizeA = n * m * sizeof(float);
	A.height = n;
	A.width = m;
	A.elements = new float[sizeA];

	int sizeB = m * q * sizeof(float);
	B.height = m;
	B.width = q;
	B.elements = new float[sizeB];

	int sizeC = n * q * sizeof(float);
	C.height = n;
	C.width = q;
	C.elements = new float[sizeC];

	srand(time(NULL));
	for (int i = 0; i < A.height * A.width; i++) {
		A.elements[i] = rand() % 10;
	}

	for (int i = 0; i < B.height * B.width; i++) {
		B.elements[i] = rand() % 10;
	}

	//print(A);
	//printf("\n");
	//print(B);
	//printf("\n");

	MatMul(A, B, C);

	free(A.elements);
	free(B.elements);
	free(C.elements);

	return 0;
}

// Matrix multiplication kernel called by MatMul()
__global__ void MatMulKernel(Matrix A, Matrix B, Matrix C) {
// Each thread computes one element of C
// by accumulating results into Cvalue
	float Cvalue = 0;
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int col = blockIdx.x * blockDim.x + threadIdx.x;
	for (int e = 0; e < A.width; ++e)
		Cvalue += A.elements[row * A.width + e] * B.elements[e * B.width + col];
	C.elements[row * C.width + col] = Cvalue;
}
