#include "hip/hip_runtime.h"
/**
 * Copyright 1993-2012 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 */
#include <stdio.h>
#include <stdlib.h>
#include <iostream>

#include <hip/hip_runtime.h>

// Matrices are stored in row-major order:
// M(row, col) = *(M.elements + row * M.width + col)
typedef struct {
	int width;
	int height;
	float* elements;
} Matrix;

// Thread block size
#ifndef BLOCK_SIZE
#define BLOCK_SIZE 16
#endif

// Forward declaration of the matrix multiplication kernel
__global__ void MatMulKernel(const Matrix, const Matrix, Matrix);

void print(Matrix X) {
//	std::cout << X.height << "\n"; std::cout << X.width << "\n";
	for (int i = 0; i < X.height; i++) {
		for (int j = 0; j < X.width; j++) {
			std::cout << X.elements[i * X.width + j] << " ";
		}
		std::cout << "\n";
	}
}

// Matrix multiplication - Host code
// Matrix dimensions are assumed to be multiples of BLOCK_SIZE
void MatMul(const Matrix A, const Matrix B, Matrix C) {
// Load A and B to device memory
	Matrix d_A;
	d_A.width = A.width;
	d_A.height = A.height;
	size_t size = A.width * A.height * sizeof(float);
	hipMalloc(&d_A.elements, size);
	hipMemcpy(d_A.elements, A.elements, size, hipMemcpyHostToDevice);
	Matrix d_B;
	d_B.width = B.width;
	d_B.height = B.height;
	size = B.width * B.height * sizeof(float);
	hipMalloc(&d_B.elements, size);
	hipMemcpy(d_B.elements, B.elements, size, hipMemcpyHostToDevice);
// Allocate C in device memory
	Matrix d_C;
	d_C.width = C.width;
	d_C.height = C.height;
	size = C.width * C.height * sizeof(float);
	hipMalloc(&d_C.elements, size);
// Invoke kernel
	//dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
	dim3 dimBlock(BLOCK_SIZE, 1, 1);
	//dim3 dimGrid((B.width - 1) / dimBlock.x + 1, (A.height - 1) / dimBlock.y + 1);
	dim3 dimGrid((B.width - 1) / dimBlock.x + 1, A.height);

	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	hipEventRecord(start);
	MatMulKernel<<<dimGrid, dimBlock>>>(d_A, d_B, d_C);
	hipEventRecord(stop);

	hipError_t errSync = hipGetLastError();
	hipError_t errAsync = hipDeviceSynchronize();
	if (errSync != hipSuccess)
		printf("4: Sync kernel error: %s\n", hipGetErrorString(errSync));
	if (errAsync != hipSuccess)
		printf("4: Async kernel error: %s\n", hipGetErrorString(errAsync));

	// Read C from device memory
	hipMemcpy(C.elements, d_C.elements, size, hipMemcpyDeviceToHost);

	if (ELAPSED_TIME == 1) {
		hipEventSynchronize (stop);
		float milliseconds = 0;
		hipEventElapsedTime(&milliseconds, start, stop);
		std::cout << milliseconds << "\n";
	}
	else {
		print(C);
	}

// Free device memory
	hipFree(d_A.elements);
	hipFree(d_B.elements);
	hipFree(d_C.elements);
}

int main() {

	int n, m, q;

	scanf("%d", &n);
	m = n;
	q = n;
	//printf("n=%d,m=%d,q=%d\n", n, m, q);

	Matrix A;
	Matrix B;
	Matrix C;

	int sizeA = n * m * sizeof(float);
	A.height = n;
	A.width = m;
	A.elements = new float[sizeA];

	int sizeB = m * q * sizeof(float);
	B.height = m;
	B.width = q;
	B.elements = new float[sizeB];

	int sizeC = n * q * sizeof(float);
	C.height = n;
	C.width = q;
	C.elements = new float[sizeC];

	srand(time(NULL));
	for (int i = 0; i < n*m; i++)
		scanf("%f", &A.elements[i]);

	for (int i = 0; i < m*q; i++)
		scanf("%f", &B.elements[i]);

	//print(A);
	//printf("\n");
	//print(B);
	//printf("\n");

	MatMul(A, B, C);

	free(A.elements);
	free(B.elements);
	free(C.elements);

	return 0;
}

// Matrix multiplication kernel called by MatMul()
__global__ void MatMulKernel(Matrix A, Matrix B, Matrix C) {
// Each thread computes one element of C
// by accumulating results into Cvalue
	float Cvalue = 0;
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int col = blockIdx.x * blockDim.x + threadIdx.x;

	for (int e = 0; e < A.width; ++e)
		Cvalue += A.elements[row * A.width + e] * B.elements[e * B.width + col];

	C.elements[row * C.width + col] = Cvalue;
}
