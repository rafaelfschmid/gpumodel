#include "hip/hip_runtime.h"
/**
 * Copyright 1993-2012 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 */
#include <stdio.h>
#include <stdlib.h>
#include <iostream>

#include <hip/hip_runtime.h>

// Matrices are stored in row-major order:
// M(row, col) = *(M.elements + row * M.stride + col)
typedef struct {
	int width;
	int height;
	int stride;
	float* elements;
} Matrix;

// Thread block size
#ifndef BLOCK_SIZE
#define BLOCK_SIZE 16
#endif

// Get a matrix element
__device__ float GetElement(const Matrix A, int row, int col) {
	return A.elements[row * A.stride + col];
}

// Set a matrix element
__device__ void SetElement(Matrix A, int row, int col, float value) {
	A.elements[row * A.stride + col] = value;
}

// Get the BLOCK_SIZExBLOCK_SIZE sub-matrix Asub of A that is
// located col sub-matrices to the right and row sub-matrices down
// from the upper-left corner of A
__device__ Matrix GetSubMatrix(Matrix A, int row, int col) {
	Matrix Asub;
	Asub.width = BLOCK_SIZE;
	Asub.height = BLOCK_SIZE;
	Asub.stride = A.stride;
	Asub.elements = &A.elements[A.stride * BLOCK_SIZE * row + BLOCK_SIZE * col];
	return Asub;
}

// Forward declaration of the matrix multiplication kernel
__global__ void MatMulKernel(const Matrix, const Matrix, Matrix);

void print(Matrix X) {
	for (int i = 0; i < X.height; i++) {
		for (int j = 0; j < X.width; j++) {
			std::cout << X.elements[i * X.width + j] << " ";
		}
		std::cout << "\n";
	}
}

// Matrix multiplication - Host code
// Matrix dimensions are assumed to be multiples of BLOCK_SIZE
void MatMul(const Matrix A, const Matrix B, Matrix C) {
	// Load A and B to device memory
	Matrix d_A;
	d_A.width = d_A.stride = A.width;
	d_A.height = A.height;
	size_t size = A.width * A.height * sizeof(float);
	hipMalloc(&d_A.elements, size);
	hipMemcpy(d_A.elements, A.elements, size, hipMemcpyHostToDevice);
	Matrix d_B;
	d_B.width = d_B.stride = B.width;
	d_B.height = B.height;
	size = B.width * B.height * sizeof(float);

	hipMalloc(&d_B.elements, size);
	hipMemcpy(d_B.elements, B.elements, size, hipMemcpyHostToDevice);
// Allocate C in device memory
	Matrix d_C;
	d_C.width = d_C.stride = C.width;
	d_C.height = C.height;
	size = C.width * C.height * sizeof(float);
	hipMalloc(&d_C.elements, size);
// Invoke kernel
	dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
	//dim3 dimGrid(B.width / dimBlock.x, A.height / dimBlock.y);
	dim3 dimGrid((B.width - 1) / dimBlock.x + 1,
			(A.height - 1) / dimBlock.y + 1);

	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	hipEventRecord(start);
	MatMulKernel<<<dimGrid, dimBlock>>>(d_A, d_B, d_C);
	hipEventRecord(stop);

	hipError_t errSync = hipGetLastError();
	hipError_t errAsync = hipDeviceSynchronize();
	if (errSync != hipSuccess)
		printf("4: Sync kernel error: %s\n", hipGetErrorString(errSync));
	if (errAsync != hipSuccess)
		printf("4: Async kernel error: %s\n", hipGetErrorString(errAsync));

	// Read C from device memory
	hipMemcpy(C.elements, d_C.elements, size, hipMemcpyDeviceToHost);

	if (ELAPSED_TIME == 1) {
		hipEventSynchronize(stop);
		float milliseconds = 0;
		hipEventElapsedTime(&milliseconds, start, stop);
		std::cout << milliseconds << "\n";
	} else {
		print(C);
	}

// Free device memory
	hipFree(d_A.elements);
	hipFree(d_B.elements);
	hipFree(d_C.elements);
}

int main() {

	int n, m, q;

	scanf("%d", &n);
	m = q = n;

	Matrix A;
	Matrix B;
	Matrix C;

	int sizeA = n * m * sizeof(float);
	A.height = n;
	A.width = m;
	A.elements = new float[sizeA];

	int sizeB = m * q * sizeof(float);
	B.height = m;
	B.width = q;
	B.elements = new float[sizeB];

	int sizeC = n * q * sizeof(float);
	C.height = n;
	C.width = q;
	C.elements = new float[sizeC];

	srand (time(NULL));
	for(int i = 0; i < n*m; i++)
		scanf("%f", &A.elements[i]);

	for (int i = 0; i < m * q; i++)
		scanf("%f", &B.elements[i]);

	//print(A);
	//printf("\n");
	//print(B);
	//printf("\n");

	MatMul(A, B, C);

	free(A.elements);
	free(B.elements);
	free(C.elements);

	return 0;
}

// Matrix multiplication kernel called by MatMul()
__global__ void MatMulKernel(Matrix A, Matrix B, Matrix C) {
// Block row and column
	int blockRow = blockIdx.y;
	int blockCol = blockIdx.x;
// Each thread block computes one sub-matrix Csub of C
	Matrix Csub = GetSubMatrix(C, blockRow, blockCol);
// Each thread computes one element of Csub
// by accumulating results into Cvalue
	float Cvalue = 0;
// Thread row and column within Csub
	int row = threadIdx.y;
	int col = threadIdx.x;

	// Loop over all the sub-matrices of A and B that are
	// required to compute Csub
	// Multiply each pair of sub-matrices together
	// and accumulate the results
	for (int m = 0; m < (A.width / BLOCK_SIZE); ++m) {
		// Get sub-matrix Asub of A
		Matrix Asub = GetSubMatrix(A, blockRow, m);
		// Get sub-matrix Bsub of B
		Matrix Bsub = GetSubMatrix(B, m, blockCol);

		// Multiply Asub and Bsub together
		for (int e = 0; e < BLOCK_SIZE; ++e) {
			Cvalue += GetElement(Asub, row, e) * GetElement(Bsub, e, col);;
		}
		// Synchronize to make sure that the preceding
		// computation is done before loading two new
		// sub-matrices of A and B in the next iteration
		//__syncthreads();
	}

	// Write Csub to device memory
	// Each thread writes one element
	SetElement(Csub, row, col, Cvalue);
}
